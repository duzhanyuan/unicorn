
/**
 * Copyright (c) 2011 Indian Institute of Technology, New Delhi
 * All Rights Reserved
 *
 * Entire information in this file and PMLIB software is property
 * of Indian Institue of Technology, New Delhi. Redistribution, 
 * modification and any use in source form is strictly prohibited
 * without formal written approval from Indian Institute of Technology, 
 * New Delhi. Use of software in binary form is allowed provided
 * the using application clearly highlights the credits.
 *
 * This work is the doctoral project of Tarun Beri under the guidance
 * of Prof. Subodh Kumar and Prof. Sorav Bansal. More information
 * about the authors is available at their websites -
 * Prof. Subodh Kumar - http://www.cse.iitd.ernet.in/~subodh/
 * Prof. Sorav Bansal - http://www.cse.iitd.ernet.in/~sbansal/
 * Tarun Beri - http://www.cse.iitd.ernet.in/~tarun
 */

#include "pmBase.h"
#include "pmDispatcherGPU.h"
#include "pmHardware.h"
#include "pmMemSection.h"

#ifdef SUPPORT_CUDA

#include "pmLogger.h"
#include <string>

#endif

namespace pm
{

using namespace dispatcherCUDA;

#ifdef SUPPORT_CUDA

hipError_t (*gFuncPtr_cudaGetDeviceCount)(int* count);
hipError_t (*gFuncPtr_cudaGetDeviceProperties)(struct hipDeviceProp_t* prop, int device);
hipError_t (*gFuncPtr_cudaSetDevice)(int device);
hipError_t (*gFuncPtr_cudaMalloc)(void** pCudaPtr, int pLength);
hipError_t (*gFuncPtr_cudaMemcpy)(void* pCudaPtr, void* pHostPtr, int pLength, int pDirection);
hipError_t (*gFuncPtr_cudaFree)(void* pCudaPtr);


#define EXECUTE_CUDA_SYMBOL(libPtr, symbol, prototype, ...) \
	{ \
		void* dSymbolPtr = GetExportedSymbol(libPtr, symbol); \
		if(!dSymbolPtr)	\
		{ \
			std::string dStr("Undefined CUDA Symbol "); \
			dStr += symbol; \
			pmLogger::GetLogger()->Log(pmLogger::DEBUG_INTERNAL, pmLogger::ERROR, dStr.c_str()); \
			PMTHROW(pmExceptionGPU(pmExceptionGPU::NVIDIA_CUDA, pmExceptionGPU::UNDEFINED_SYMBOL)); \
		} \
		*(void**)(&prototype) = dSymbolPtr; \
		(*prototype)(__VA_ARGS__); \
	}

#define SAFE_EXECUTE_CUDA(libPtr, symbol, prototype, ...) \
	{ \
		EXECUTE_CUDA_SYMBOL(libPtr, symbol, prototype, __VA_ARGS__); \
		hipError_t dErrorCUDA = hipGetLastError(); \
		if(dErrorCUDA != hipSuccess) \
		{ \
			pmLogger::GetLogger()->Log(pmLogger::MINIMAL, pmLogger::ERROR, hipGetErrorString(dErrorCUDA)); \
			PMTHROW(pmExceptionGPU(pmExceptionGPU::NVIDIA_CUDA, pmExceptionGPU::RUNTIME_ERROR)); \
		} \
	}

pmStatus pmDispatcherCUDA::CountAndProbeProcessingElements()
{
	int lCountCUDA = 0;
	mCountCUDA = 0;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipGetDeviceCount", gFuncPtr_cudaGetDeviceCount, &lCountCUDA );

	for(int i = 0; i<lCountCUDA; ++i)
	{
		hipDeviceProp_t lDeviceProp;
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipGetDeviceProperties", gFuncPtr_cudaGetDeviceProperties, &lDeviceProp, i );

		if(!(lDeviceProp.major == 9999 && lDeviceProp.minor == 9999))
			mDeviceVector.push_back(std::pair<int, hipDeviceProp_t>(i, lDeviceProp));			
	}

	mCountCUDA = mDeviceVector.size();

	return pmSuccess;
}

pmStatus pmDispatcherCUDA::BindToDevice(size_t pDeviceIndex)
{
	int lHardwareId = mDeviceVector[pDeviceIndex].first;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipSetDevice", gFuncPtr_cudaSetDevice, lHardwareId );

	return pmSuccess;
}

std::string pmDispatcherCUDA::GetDeviceName(size_t pDeviceIndex)
{
	hipDeviceProp_t lProp = mDeviceVector[pDeviceIndex].second;
	return lProp.name;
}

std::string pmDispatcherCUDA::GetDeviceDescription(size_t pDeviceIndex)
{
	hipDeviceProp_t lProp = mDeviceVector[pDeviceIndex].second;
	std::string lStr("Clock Rate=");
	lStr += lProp.clockRate;
	lStr += ";sharedMemPerBlock=";
	lStr += lProp.sharedMemPerBlock;

	return lStr;
}

pmStatus pmDispatcherCUDA::InvokeKernel(size_t pBoundDeviceIndex, pmTaskInfo& pTaskInfo, pmSubtaskInfo& pSubtaskInfo, pmCudaLaunchConf& pCudaLaunchConf, bool pOutputMemWriteOnly, pmSubtaskCallback_GPU_CUDA pKernelPtr, uint pOriginatingMachineIndex, ulong pSequenceNumber, pmMemSection* pInputMemSection)
{
    pmMachine* lOriginatingHost = pmMachinePool::GetMachinePool()->GetMachine(pOriginatingMachineIndex);
    pmTask* lTask = pmTaskManager::GetTaskManager()->FindTask(lOriginatingHost, pSequenceNumber);

    bool lMatchingLastExecutionRecord = false;
    lastExecutionRecord* lLastRecord = NULL;

    // Auto lock/unlock scope
    {
        FINALIZE_RESOURCE_PTR(dLastExecutionLock, RESOURCE_LOCK_IMPLEMENTATION_CLASS, &mLastExecutionLock, Lock(), Unlock());
        if(mLastExecutionMap.find(pBoundDeviceIndex) != mLastExecutionMap.end())
            lLastRecord = &(mLastExecutionMap[pBoundDeviceIndex]);
    }

    if(lLastRecord && lLastRecord->taskOriginatingMachineIndex == pOriginatingMachineIndex && lLastRecord->taskSequenceNumber == pSequenceNumber)
        lMatchingLastExecutionRecord = true;
    
    void* lTaskConfCudaPtr = NULL;
    if(pTaskInfo.taskConf && pTaskInfo.taskConfLength != 0)
    {
        if(lMatchingLastExecutionRecord && lLastRecord->taskConfCudaPtr)
        {
            lTaskConfCudaPtr = lLastRecord->taskConfCudaPtr;
        }
        else
        {
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lTaskConfCudaPtr, pTaskInfo.taskConfLength );
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lTaskConfCudaPtr, pTaskInfo.taskConf, pTaskInfo.taskConfLength, hipMemcpyHostToDevice );
        }
    }

    void* lInputMemCudaPtr = NULL;
    void* lOutputMemCudaPtr = NULL;

    if(pSubtaskInfo.inputMem && pSubtaskInfo.inputMemLength != 0)
    {
        if(lMatchingLastExecutionRecord && lTask->GetSubscriptionManager().SubtasksHaveMatchingSubscriptions(lLastRecord->lastSubtaskId, pSubtaskInfo.subtaskId, true))
        {
            lInputMemCudaPtr = lLastRecord->inputMemCudaPtr;
        }
        else
        {
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lInputMemCudaPtr, pSubtaskInfo.inputMemLength );
            
            subscription::subscriptionRecordType::iterator lBegin, lEnd;
            lTask->GetSubscriptionManager().GetNonConsolidatedInputMemSubscriptionsForSubtask(pSubtaskInfo.subtaskId, lBegin, lEnd);
            if(lBegin == lEnd)
            {
                SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lInputMemCudaPtr, pSubtaskInfo.inputMem, pSubtaskInfo.inputMemLength, hipMemcpyHostToDevice );
            }
            else
            {
                void* lTempDevicePtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(lInputMemCudaPtr) + lBegin->first);
                void* lTempHostPtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(pSubtaskInfo.inputMem) + lBegin->first);
                SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lTempDevicePtr, lTempHostPtr, lBegin->second.first, hipMemcpyHostToDevice );
            }
        }
    }
        
    if(lLastRecord && lLastRecord->inputMemCudaPtr && lLastRecord->inputMemCudaPtr != lInputMemCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord->inputMemCudaPtr );
        
    if(lLastRecord && lLastRecord->taskConfCudaPtr && lLastRecord->taskConfCudaPtr != lTaskConfCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord->taskConfCudaPtr );

    // Auto lock/unlock scope
    {
        FINALIZE_RESOURCE_PTR(dLastExecutionLock, RESOURCE_LOCK_IMPLEMENTATION_CLASS, &mLastExecutionLock, Lock(), Unlock());
        mLastExecutionMap[pBoundDeviceIndex].taskOriginatingMachineIndex = pOriginatingMachineIndex;
        mLastExecutionMap[pBoundDeviceIndex].taskSequenceNumber = pSequenceNumber;
        mLastExecutionMap[pBoundDeviceIndex].lastSubtaskId = pSubtaskInfo.subtaskId;
        mLastExecutionMap[pBoundDeviceIndex].inputMemCudaPtr = lInputMemCudaPtr;
        mLastExecutionMap[pBoundDeviceIndex].taskConfCudaPtr = lTaskConfCudaPtr;
    }

	if(pSubtaskInfo.outputMem && pSubtaskInfo.outputMemLength != 0)
	{
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lOutputMemCudaPtr, pSubtaskInfo.outputMemLength );

        if(!pOutputMemWriteOnly)
        {
            subscription::subscriptionRecordType::iterator lBegin, lEnd;
            lTask->GetSubscriptionManager().GetNonConsolidatedOutputMemSubscriptionsForSubtask(pSubtaskInfo.subtaskId, lBegin, lEnd);
            if(lBegin == lEnd)
            {
                SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lOutputMemCudaPtr, pSubtaskInfo.outputMem, pSubtaskInfo.outputMemLength, hipMemcpyHostToDevice );
            }
            else
            {
                void* lTempDevicePtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(lOutputMemCudaPtr) + lBegin->first);
                void* lTempHostPtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(pSubtaskInfo.outputMem) + lBegin->first);
                SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lTempDevicePtr, lTempHostPtr, lBegin->second.first, hipMemcpyHostToDevice );
            }
        }
	}

	pmStatus lStatus = pmStatusUnavailable;
	pmStatus* lStatusPtr = NULL;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lStatusPtr, sizeof(pmStatus) );
	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lStatusPtr, &lStatus, sizeof(pmStatus), hipMemcpyHostToDevice );

	pmTaskInfo lTaskInfo = pTaskInfo;
	lTaskInfo.taskConf = lTaskConfCudaPtr;

	pmSubtaskInfo lSubtaskInfo = pSubtaskInfo;
	lSubtaskInfo.inputMem = lInputMemCudaPtr;
	lSubtaskInfo.outputMem = lOutputMemCudaPtr;

	dim3 gridConf(pCudaLaunchConf.blocksX, pCudaLaunchConf.blocksY, pCudaLaunchConf.blocksZ);
	dim3 blockConf(pCudaLaunchConf.threadsX, pCudaLaunchConf.threadsY, pCudaLaunchConf.threadsZ);

	//pKernelPtr <<<gridConf, blockConf, pCudaLaunchConf.sharedMem>>> (pTaskInfo, lSubtaskInfo, lStatusPtr);
	pKernelPtr <<<gridConf, blockConf>>> (lTaskInfo, lSubtaskInfo, lStatusPtr);

	if(hipGetLastError() == hipSuccess)
	{
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, pSubtaskInfo.outputMem, lOutputMemCudaPtr, pSubtaskInfo.outputMemLength, hipMemcpyDeviceToHost );
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, &lStatus, lStatusPtr, sizeof(pmStatus), hipMemcpyDeviceToHost );
	}

	if(lOutputMemCudaPtr)
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lOutputMemCudaPtr );

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lStatusPtr );

	return lStatus;
}

pmStatus pmDispatcherCUDA::FreeLastExecutionResources(size_t pBoundDeviceIndex)
{
    FINALIZE_RESOURCE_PTR(dLastExecutionLock, RESOURCE_LOCK_IMPLEMENTATION_CLASS, &mLastExecutionLock, Lock(), Unlock());

    if(mLastExecutionMap.find(pBoundDeviceIndex) == mLastExecutionMap.end())
        return pmSuccess;
        
    lastExecutionRecord& lLastRecord = mLastExecutionMap[pBoundDeviceIndex];
	
    if(lLastRecord.inputMemCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord.inputMemCudaPtr );
        
    if(lLastRecord.taskConfCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord.taskConfCudaPtr );
    
    mLastExecutionMap.erase(pBoundDeviceIndex);
    
    return pmSuccess;
}

#else	// SUPPORT_CUDA
/* The below functions are there to satisfy compiler. These are never executed. */
pmStatus pmDispatcherCUDA::CountAndProbeProcessingElements()
{
	mCountCUDA = 0;
	return pmSuccess;
}

pmStatus pmDispatcherCUDA::BindToDevice(size_t pDeviceIndex)
{
	return pmSuccess;
}

std::string pmDispatcherCUDA::GetDeviceName(size_t pDeviceIndex)
{
	return std::string();
}

std::string pmDispatcherCUDA::GetDeviceDescription(size_t pDeviceIndex)
{
	return std::string();
}

#endif	// SUPPORT_CUDA

}
