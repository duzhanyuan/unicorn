
/**
 * Copyright (c) 2011 Indian Institute of Technology, New Delhi
 * All Rights Reserved
 *
 * Entire information in this file and PMLIB software is property
 * of Indian Institue of Technology, New Delhi. Redistribution, 
 * modification and any use in source form is strictly prohibited
 * without formal written approval from Indian Institute of Technology, 
 * New Delhi. Use of software in binary form is allowed provided
 * the using application clearly highlights the credits.
 *
 * This work is the doctoral project of Tarun Beri under the guidance
 * of Prof. Subodh Kumar and Prof. Sorav Bansal. More information
 * about the authors is available at their websites -
 * Prof. Subodh Kumar - http://www.cse.iitd.ernet.in/~subodh/
 * Prof. Sorav Bansal - http://www.cse.iitd.ernet.in/~sbansal/
 * Tarun Beri - http://www.cse.iitd.ernet.in/~tarun
 */

#include "pmBase.h"
#include "pmDispatcherGPU.h"
#include <string>

#ifdef SUPPORT_CUDA
#include "pmLogger.h"
#endif

namespace pm
{

using namespace dispatcherCUDA;

#ifdef SUPPORT_CUDA

hipError_t (*gFuncPtr_cudaGetDeviceCount)(int* count);
hipError_t (*gFuncPtr_cudaGetDeviceProperties)(struct hipDeviceProp_t* prop, int device);
hipError_t (*gFuncPtr_cudaSetDevice)(int device);
hipError_t (*gFuncPtr_cudaMalloc)(void** pCudaPtr, int pLength);
hipError_t (*gFuncPtr_cudaMemcpy)(void* pCudaPtr, void* pHostPtr, int pLength, int pDirection);
hipError_t (*gFuncPtr_cudaFree)(void* pCudaPtr);


#define EXECUTE_CUDA_SYMBOL(libPtr, symbol, prototype, ...) \
	{ \
		void* dSymbolPtr = GetExportedSymbol(libPtr, symbol); \
		if(!dSymbolPtr)	\
		{ \
			std::string dStr("Undefined CUDA Symbol "); \
			dStr += symbol; \
			pmLogger::GetLogger()->Log(pmLogger::DEBUG_INTERNAL, pmLogger::ERROR, dStr.c_str()); \
			PMTHROW(pmExceptionGPU(pmExceptionGPU::NVIDIA_CUDA, pmExceptionGPU::UNDEFINED_SYMBOL)); \
		} \
		*(void**)(&prototype) = dSymbolPtr; \
		(*prototype)(__VA_ARGS__); \
	}

#define SAFE_EXECUTE_CUDA(libPtr, symbol, prototype, ...) \
	{ \
		EXECUTE_CUDA_SYMBOL(libPtr, symbol, prototype, __VA_ARGS__); \
		hipError_t dErrorCUDA = hipGetLastError(); \
		if(dErrorCUDA != hipSuccess) \
		{ \
			pmLogger::GetLogger()->Log(pmLogger::MINIMAL, pmLogger::ERROR, hipGetErrorString(dErrorCUDA)); \
			PMTHROW(pmExceptionGPU(pmExceptionGPU::NVIDIA_CUDA, pmExceptionGPU::RUNTIME_ERROR)); \
		} \
	}

pmStatus pmDispatcherCUDA::CountAndProbeProcessingElements()
{
	int lCountCUDA = 0;
	mCountCUDA = 0;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipGetDeviceCount", gFuncPtr_cudaGetDeviceCount, &lCountCUDA );

	for(int i = 0; i<lCountCUDA; ++i)
	{
		hipDeviceProp_t lDeviceProp;
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipGetDeviceProperties", gFuncPtr_cudaGetDeviceProperties, &lDeviceProp, i );

		if(!(lDeviceProp.major == 9999 && lDeviceProp.minor == 9999))
			mDeviceVector.push_back(std::pair<int, hipDeviceProp_t>(i, lDeviceProp));			
	}

	mCountCUDA = mDeviceVector.size();

	return pmSuccess;
}

pmStatus pmDispatcherCUDA::BindToDevice(size_t pDeviceIndex)
{
	int lHardwareId = mDeviceVector[pDeviceIndex].first;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipSetDevice", gFuncPtr_cudaSetDevice, lHardwareId );

	return pmSuccess;
}

std::string pmDispatcherCUDA::GetDeviceName(size_t pDeviceIndex)
{
	hipDeviceProp_t lProp = mDeviceVector[pDeviceIndex].second;
	return lProp.name;
}

std::string pmDispatcherCUDA::GetDeviceDescription(size_t pDeviceIndex)
{
	hipDeviceProp_t lProp = mDeviceVector[pDeviceIndex].second;
	std::string lStr("Clock Rate=");
	lStr += lProp.clockRate;
	lStr += ";sharedMemPerBlock=";
	lStr += lProp.sharedMemPerBlock;

	return lStr;
}
    
void* pmDispatcherCUDA::GetDeviceInfoCudaPtr(pmDeviceInfo& pDeviceInfo)
{
    void* lDeviceInfoCudaPtr = NULL;

    SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lDeviceInfoCudaPtr, sizeof(pDeviceInfo) );
    SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lDeviceInfoCudaPtr, &pDeviceInfo, sizeof(pDeviceInfo), hipMemcpyHostToDevice );
    
    return lDeviceInfoCudaPtr;
}
    
void pmDispatcherCUDA::FreeDeviceInfoCudaPtr(void* pDeviceInfoCudaPtr)
{
    SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, pDeviceInfoCudaPtr );
}
    
class pmCudaAutoPtr : public pmBase
{
public:
    pmCudaAutoPtr(void* pRuntimeHandle, size_t pAllocationSize = 0)
    : mRuntimeHandle(pRuntimeHandle)
    , mCudaPtr(NULL)
    {
        if(pAllocationSize)
        {
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&mCudaPtr, pAllocationSize );
        }
    }
    
    ~pmCudaAutoPtr()
    {
        if(mCudaPtr)
        {
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, mCudaPtr );
        }
    }
    
    void reset(size_t pAllocationSize)
    {
        if(mCudaPtr)
        {
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, mCudaPtr );
        }

        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&mCudaPtr, pAllocationSize );
    }
    
    void release()
    {
        mCudaPtr = NULL;
    }
    
    void* getPtr()
    {
        return mCudaPtr;
    }
    
private:
    void* mRuntimeHandle;
    void* mCudaPtr;
};

pmStatus pmDispatcherCUDA::InvokeKernel(pmExecutionStub* pStub, size_t pBoundDeviceIndex, pmTaskInfo& pTaskInfo, pmDeviceInfo& pDeviceInfo, void* pDeviceInfoCudaPtr, pmSubtaskInfo& pSubtaskInfo, pmCudaLaunchConf& pCudaLaunchConf, bool pOutputMemWriteOnly, pmSubtaskCallback_GPU_CUDA pKernelPtr, pmSubtaskCallback_GPU_Custom pCustomKernelPtr, uint pOriginatingMachineIndex, ulong pSequenceNumber, void* pTaskOutputMem)
{
    pmCudaAutoPtr lInputMemAutoPtr(mRuntimeHandle), lOutputMemAutoPtr(mRuntimeHandle);
    pmCudaAutoPtr lScratchBufferAutoPtr(mRuntimeHandle), lTaskConfAutoPtr(mRuntimeHandle), lStatusAutoPtr(mRuntimeHandle);

    bool lMatchingLastExecutionRecord = false;
    lastExecutionRecord* lLastRecord = NULL;

    // Auto lock/unlock scope
    {
        FINALIZE_RESOURCE_PTR(dLastExecutionLock, RESOURCE_LOCK_IMPLEMENTATION_CLASS, &mLastExecutionLock, Lock(), Unlock());
        if(mLastExecutionMap.find(pBoundDeviceIndex) != mLastExecutionMap.end())
            lLastRecord = &(mLastExecutionMap[pBoundDeviceIndex]);
    }

    if(lLastRecord && lLastRecord->taskOriginatingMachineIndex == pOriginatingMachineIndex && lLastRecord->taskSequenceNumber == pSequenceNumber)
        lMatchingLastExecutionRecord = true;
    
    void* lTaskConfCudaPtr = NULL;
    if(pTaskInfo.taskConf && pTaskInfo.taskConfLength != 0)
    {
        if(lMatchingLastExecutionRecord && lLastRecord->taskConfCudaPtr)
        {
            lTaskConfCudaPtr = lLastRecord->taskConfCudaPtr;
        }
        else
        {
            lTaskConfAutoPtr.reset(pTaskInfo.taskConfLength);
            lTaskConfCudaPtr = lTaskConfAutoPtr.getPtr();

            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lTaskConfCudaPtr, pTaskInfo.taskConf, pTaskInfo.taskConfLength, hipMemcpyHostToDevice );
        }
    }

    void* lInputMemCudaPtr = NULL;
    void* lOutputMemCudaPtr = NULL;

    if(pSubtaskInfo.inputMem && pSubtaskInfo.inputMemLength != 0)
    {
        if(lMatchingLastExecutionRecord && SubtasksHaveMatchingSubscriptions(pStub, pOriginatingMachineIndex, pSequenceNumber, lLastRecord->lastSubtaskId, pSubtaskInfo.subtaskId, INPUT_MEM_READ_SUBSCRIPTION))
        {
            lInputMemCudaPtr = lLastRecord->inputMemCudaPtr;
        }
        else
        {
            lInputMemAutoPtr.reset(pSubtaskInfo.inputMemLength);
            lInputMemCudaPtr = lInputMemAutoPtr.getPtr();

            pmSubscriptionInfo lInputMemSubscriptionInfo;
            GetInputMemSubscriptionForSubtask(pStub, pOriginatingMachineIndex, pSequenceNumber, pSubtaskInfo, lInputMemSubscriptionInfo);

            std::vector<std::pair<size_t, size_t> > lSubscriptionVector;
            GetNonConsolidatedSubscriptionsForSubtask(pStub, pOriginatingMachineIndex, pSequenceNumber, INPUT_MEM_READ_SUBSCRIPTION, pSubtaskInfo, lSubscriptionVector);
            
            std::vector<std::pair<size_t, size_t> >::iterator lIter = lSubscriptionVector.begin(), lEndIter = lSubscriptionVector.end();
            for(; lIter != lEndIter; ++lIter)
            {
                void* lTempDevicePtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(lInputMemCudaPtr) + (*lIter).first - lInputMemSubscriptionInfo.offset);
                void* lTempHostPtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(pSubtaskInfo.inputMem) + (*lIter).first);
                SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lTempDevicePtr, lTempHostPtr, (*lIter).second, hipMemcpyHostToDevice );
            }
        }
    }
        
    if(lLastRecord && lLastRecord->inputMemCudaPtr && lLastRecord->inputMemCudaPtr != lInputMemCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord->inputMemCudaPtr );
        
    if(lLastRecord && lLastRecord->taskConfCudaPtr && lLastRecord->taskConfCudaPtr != lTaskConfCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord->taskConfCudaPtr );

    lInputMemAutoPtr.release();
    lTaskConfAutoPtr.release();

    // Auto lock/unlock scope
    {
        FINALIZE_RESOURCE_PTR(dLastExecutionLock, RESOURCE_LOCK_IMPLEMENTATION_CLASS, &mLastExecutionLock, Lock(), Unlock());
        mLastExecutionMap[pBoundDeviceIndex].taskOriginatingMachineIndex = pOriginatingMachineIndex;
        mLastExecutionMap[pBoundDeviceIndex].taskSequenceNumber = pSequenceNumber;
        mLastExecutionMap[pBoundDeviceIndex].lastSubtaskId = pSubtaskInfo.subtaskId;
        mLastExecutionMap[pBoundDeviceIndex].inputMemCudaPtr = lInputMemCudaPtr;
        mLastExecutionMap[pBoundDeviceIndex].taskConfCudaPtr = lTaskConfCudaPtr;
    }
    
    pmSubscriptionInfo lUnifiedSubscriptionInfo;
    GetUnifiedOutputMemSubscriptionForSubtask(pStub, pOriginatingMachineIndex, pSequenceNumber, pSubtaskInfo, lUnifiedSubscriptionInfo);
    
	if(pSubtaskInfo.outputMem && pSubtaskInfo.outputMemLength != 0)
	{
        lOutputMemAutoPtr.reset(pSubtaskInfo.outputMemLength);
        lOutputMemCudaPtr = lOutputMemAutoPtr.getPtr();

        if(!pOutputMemWriteOnly)
        {
            std::vector<std::pair<size_t, size_t> > lSubscriptionVector;
            GetNonConsolidatedSubscriptionsForSubtask(pStub, pOriginatingMachineIndex, pSequenceNumber, OUTPUT_MEM_READ_SUBSCRIPTION, pSubtaskInfo, lSubscriptionVector);

            std::vector<std::pair<size_t, size_t> >::iterator lIter = lSubscriptionVector.begin(), lEndIter = lSubscriptionVector.end();
            for(; lIter != lEndIter; ++lIter)
            {
                void* lTempDevicePtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(lOutputMemCudaPtr) + ((*lIter).first - lUnifiedSubscriptionInfo.offset));
                void* lTempHostPtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(pTaskOutputMem) + (*lIter).first);
                SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lTempDevicePtr, lTempHostPtr, (*lIter).second, hipMemcpyHostToDevice );
            }
        }
	}

	pmStatus lStatus = pmStatusUnavailable;

    lStatusAutoPtr.reset(sizeof(pmStatus));
    pmStatus* lStatusPtr = (pmStatus*)lStatusAutoPtr.getPtr();

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lStatusPtr, &lStatus, sizeof(pmStatus), hipMemcpyHostToDevice );

	pmTaskInfo lTaskInfo = pTaskInfo;
	lTaskInfo.taskConf = lTaskConfCudaPtr;

	pmSubtaskInfo lSubtaskInfo = pSubtaskInfo;
	lSubtaskInfo.inputMem = lInputMemCudaPtr;
	lSubtaskInfo.outputMem = lOutputMemCudaPtr;
    lSubtaskInfo.outputMemRead = lSubtaskInfo.outputMemWrite = NULL;
    lSubtaskInfo.outputMemReadLength = lSubtaskInfo.outputMemWriteLength = 0;
    if(lOutputMemCudaPtr)
    {
        if(!pOutputMemWriteOnly)
        {
            lSubtaskInfo.outputMemRead = reinterpret_cast<void*>(reinterpret_cast<size_t>(lOutputMemCudaPtr) + reinterpret_cast<size_t>(pSubtaskInfo.outputMemRead) - reinterpret_cast<size_t>(pSubtaskInfo.outputMem));
            lSubtaskInfo.outputMemReadLength = pSubtaskInfo.outputMemReadLength;
        }

        lSubtaskInfo.outputMemWrite = reinterpret_cast<void*>(reinterpret_cast<size_t>(lOutputMemCudaPtr) + reinterpret_cast<size_t>(pSubtaskInfo.outputMemWrite) - reinterpret_cast<size_t>(pSubtaskInfo.outputMem));
        lSubtaskInfo.outputMemWriteLength = pSubtaskInfo.outputMemWriteLength;
    }

    lSubtaskInfo.inputMemLength = pSubtaskInfo.inputMemLength;
    
    lSubtaskInfo.gpuContext.scratchBuffer = NULL;
    pmScratchBufferInfo lScratchBufferInfo = SUBTASK_TO_POST_SUBTASK;
    size_t lScratchBufferSize = 0;
    void* lCpuScratchBuffer = CheckAndGetScratchBuffer(pStub, pOriginatingMachineIndex, pSequenceNumber, pSubtaskInfo.subtaskId, lScratchBufferSize, lScratchBufferInfo);
    if(lCpuScratchBuffer && lScratchBufferSize)
    {
        lScratchBufferAutoPtr.reset(lScratchBufferSize);
        lSubtaskInfo.gpuContext.scratchBuffer = lScratchBufferAutoPtr.getPtr();

        if(lScratchBufferInfo == PRE_SUBTASK_TO_SUBTASK || lScratchBufferInfo == PRE_SUBTASK_TO_POST_SUBTASK)
        {
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lSubtaskInfo.gpuContext.scratchBuffer, lCpuScratchBuffer, lScratchBufferSize, hipMemcpyHostToDevice );
        }
    }

    MarkInsideUserCode(pStub, pSubtaskInfo.subtaskId);
    
    if(pKernelPtr)
    {
        dim3 gridConf(pCudaLaunchConf.blocksX, pCudaLaunchConf.blocksY, pCudaLaunchConf.blocksZ);
        dim3 blockConf(pCudaLaunchConf.threadsX, pCudaLaunchConf.threadsY, pCudaLaunchConf.threadsZ);

        if(pCudaLaunchConf.sharedMem)
            pKernelPtr <<<gridConf, blockConf, pCudaLaunchConf.sharedMem>>> (pTaskInfo, (pmDeviceInfo*)pDeviceInfoCudaPtr, lSubtaskInfo, lStatusPtr);
        else
            pKernelPtr <<<gridConf, blockConf>>> (lTaskInfo, (pmDeviceInfo*)pDeviceInfoCudaPtr, lSubtaskInfo, lStatusPtr);
    }
    else
    {
        pCustomKernelPtr(pTaskInfo, pDeviceInfo, lSubtaskInfo);
    }

    MarkInsideLibraryCode(pStub, pSubtaskInfo.subtaskId);

    hipError_t lLastError = hipGetLastError();
	if(lLastError == hipSuccess)
    {
        if(!RequiresPrematureExit(pStub, pSubtaskInfo.subtaskId))
        {
            if(pSubtaskInfo.outputMem && pSubtaskInfo.outputMemLength != 0)
            {
                std::vector<std::pair<size_t, size_t> > lSubscriptionVector;
                GetNonConsolidatedSubscriptionsForSubtask(pStub, pOriginatingMachineIndex, pSequenceNumber, OUTPUT_MEM_WRITE_SUBSCRIPTION, pSubtaskInfo, lSubscriptionVector);

                std::vector<std::pair<size_t, size_t> >::iterator lIter = lSubscriptionVector.begin(), lEndIter = lSubscriptionVector.end();
                for(; lIter != lEndIter; ++lIter)
                {
                    void* lTempDevicePtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(lOutputMemCudaPtr) + ((*lIter).first - lUnifiedSubscriptionInfo.offset));
                    void* lTempHostPtr = reinterpret_cast<void*>(reinterpret_cast<size_t>(pSubtaskInfo.outputMem) + ((*lIter).first - lUnifiedSubscriptionInfo.offset));
                    SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lTempHostPtr, lTempDevicePtr, (*lIter).second, hipMemcpyDeviceToHost );
                }
            }
            
            SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, &lStatus, lStatusPtr, sizeof(pmStatus), hipMemcpyDeviceToHost );

            if(lCpuScratchBuffer && lScratchBufferSize && (lScratchBufferInfo == SUBTASK_TO_POST_SUBTASK || lScratchBufferInfo == PRE_SUBTASK_TO_POST_SUBTASK))
            {
                SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lCpuScratchBuffer, lSubtaskInfo.gpuContext.scratchBuffer, lScratchBufferSize, hipMemcpyDeviceToHost );
            }
        }
    }
    else
    {
        // Check if the kernel is compiled for a different architecture and the GPU card has a different compute capability
        //std::cout << "CUDA Error: " << hipGetLastError(lLastError) << std::endl;
        PMTHROW(pmExceptionGPU(pmExceptionGPU::NVIDIA_CUDA, pmExceptionGPU::RUNTIME_ERROR));
    }

	return lStatus;
}

pmStatus pmDispatcherCUDA::FreeLastExecutionResources(size_t pBoundDeviceIndex)
{
    FINALIZE_RESOURCE_PTR(dLastExecutionLock, RESOURCE_LOCK_IMPLEMENTATION_CLASS, &mLastExecutionLock, Lock(), Unlock());

    if(mLastExecutionMap.find(pBoundDeviceIndex) == mLastExecutionMap.end())
        return pmSuccess;
        
    lastExecutionRecord& lLastRecord = mLastExecutionMap[pBoundDeviceIndex];
	
    if(lLastRecord.inputMemCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord.inputMemCudaPtr );
        
    if(lLastRecord.taskConfCudaPtr)
        SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lLastRecord.taskConfCudaPtr );
    
    mLastExecutionMap.erase(pBoundDeviceIndex);
    
    return pmSuccess;
}

#else	// SUPPORT_CUDA
/* The below functions are there to satisfy compiler. These are never executed. */
pmStatus pmDispatcherCUDA::CountAndProbeProcessingElements()
{
	mCountCUDA = 0;
	return pmSuccess;
}

pmStatus pmDispatcherCUDA::BindToDevice(size_t pDeviceIndex)
{
	return pmSuccess;
}

std::string pmDispatcherCUDA::GetDeviceName(size_t pDeviceIndex)
{
	return std::string();
}

std::string pmDispatcherCUDA::GetDeviceDescription(size_t pDeviceIndex)
{
	return std::string();
}

#endif	// SUPPORT_CUDA

}
