
#include "pmBase.h"
#include "pmDispatcherGPU.h"

#ifdef SUPPORT_CUDA

#include "pmLogger.h"
#include <string>

#endif

namespace pm
{

#ifdef SUPPORT_CUDA

hipError_t (*gFuncPtr_cudaGetDeviceCount)(int* count);
hipError_t (*gFuncPtr_cudaGetDeviceProperties)(struct hipDeviceProp_t* prop, int device);
hipError_t (*gFuncPtr_cudaSetDevice)(int device);
hipError_t (*gFuncPtr_cudaMalloc)(void** pCudaPtr, int pLength);
hipError_t (*gFuncPtr_cudaMemcpy)(void* pCudaPtr, void* pHostPtr, int pLength, int pDirection);
hipError_t (*gFuncPtr_cudaFree)(void* pCudaPtr);


#define EXECUTE_CUDA_SYMBOL(libPtr, symbol, prototype, ...) \
	{ \
		void* dSymbolPtr = GetExportedSymbol(libPtr, symbol); \
		if(!dSymbolPtr)	\
		{ \
			std::string dStr("Undefined CUDA Symbol "); \
			dStr += symbol; \
			pmLogger::GetLogger()->Log(pmLogger::DEBUG_INTERNAL, pmLogger::ERROR, dStr.c_str()); \
			PMTHROW(pmExceptionGPU(pmExceptionGPU::NVIDIA_CUDA, pmExceptionGPU::UNDEFINED_SYMBOL)); \
		} \
		*(void**)(&prototype) = dSymbolPtr; \
		(*prototype)(__VA_ARGS__); \
	}

#define SAFE_EXECUTE_CUDA(libPtr, symbol, prototype, ...) \
	{ \
		EXECUTE_CUDA_SYMBOL(libPtr, symbol, prototype, __VA_ARGS__); \
		hipError_t dErrorCUDA = hipGetLastError(); \
		if(dErrorCUDA != hipSuccess) \
		{ \
			pmLogger::GetLogger()->Log(pmLogger::MINIMAL, pmLogger::ERROR, hipGetErrorString(dErrorCUDA)); \
			PMTHROW(pmExceptionGPU(pmExceptionGPU::NVIDIA_CUDA, pmExceptionGPU::RUNTIME_ERROR)); \
		} \
	}

pmStatus pmDispatcherCUDA::CountAndProbeProcessingElements()
{
	int lCountCUDA = 0;
	mCountCUDA = 0;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipGetDeviceCount", gFuncPtr_cudaGetDeviceCount, &lCountCUDA );

	for(int i = 0; i<lCountCUDA; ++i)
	{
		hipDeviceProp_t lDeviceProp;
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipGetDeviceProperties", gFuncPtr_cudaGetDeviceProperties, &lDeviceProp, i );

		if(!(lDeviceProp.major == 9999 && lDeviceProp.minor == 9999))
			mDeviceVector.push_back(std::pair<int, hipDeviceProp_t>(i, lDeviceProp));			
	}

	mCountCUDA = mDeviceVector.size();

	return pmSuccess;
}

pmStatus pmDispatcherCUDA::BindToDevice(size_t pDeviceIndex)
{
	int lHardwareId = mDeviceVector[pDeviceIndex].first;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipSetDevice", gFuncPtr_cudaSetDevice, lHardwareId );

	return pmSuccess;
}

std::string pmDispatcherCUDA::GetDeviceName(size_t pDeviceIndex)
{
	hipDeviceProp_t lProp = mDeviceVector[pDeviceIndex].second;
	return lProp.name;
}

std::string pmDispatcherCUDA::GetDeviceDescription(size_t pDeviceIndex)
{
	hipDeviceProp_t lProp = mDeviceVector[pDeviceIndex].second;
	std::string lStr("Clock Rate=");
	lStr += lProp.clockRate;
	lStr += ";sharedMemPerBlock=";
	lStr += lProp.sharedMemPerBlock;

	return lStr;
}

pmStatus pmDispatcherCUDA::InvokeKernel(pmTaskInfo& pTaskInfo, pmSubtaskInfo& pSubtaskInfo, pmCudaLaunchConf& pCudaLaunchConf, pmSubtaskCallback_GPU_CUDA pKernelPtr)
{
	void* lInputMemCudaPtr = NULL;
	void* lOutputMemCudaPtr = NULL;

	if(pSubtaskInfo.inputMem && pSubtaskInfo.inputMemLength != 0)
	{
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lInputMemCudaPtr, pSubtaskInfo.inputMemLength );
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lInputMemCudaPtr, pSubtaskInfo.inputMem, pSubtaskInfo.inputMemLength, hipMemcpyHostToDevice );
	}

	if(pSubtaskInfo.outputMem && pSubtaskInfo.outputMemLength != 0)
	{
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lOutputMemCudaPtr, pSubtaskInfo.outputMemLength );
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lOutputMemCudaPtr, pSubtaskInfo.outputMem, pSubtaskInfo.outputMemLength, hipMemcpyHostToDevice );
	}

	pmStatus lStatus = pmStatusUnavailable;
	pmStatus* lStatusPtr = NULL;

	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMalloc", gFuncPtr_cudaMalloc, (void**)&lStatusPtr, sizeof(pmStatus) );
	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, lStatusPtr, &lStatus, sizeof(pmStatus), hipMemcpyHostToDevice );

	pmSubtaskInfo lSubtaskInfo = pSubtaskInfo;
	lSubtaskInfo.inputMem = lInputMemCudaPtr;
	lSubtaskInfo.outputMem = lOutputMemCudaPtr;

    dim3 gridConf(pCudaLaunchConf.blocksX, pCudaLaunchConf.blocksY, pCudaLaunchConf.blocksZ);
    dim3 blockConf(pCudaLaunchConf.threadsX, pCudaLaunchConf.threadsY, pCudaLaunchConf.threadsZ);

	pKernelPtr <<<gridConf, blockConf, pCudaLaunchConf.sharedMem>>> (pTaskInfo, lSubtaskInfo, lStatusPtr);

	if(hipGetLastError() == hipSuccess)
	{
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, pSubtaskInfo.outputMem, lOutputMemCudaPtr, pSubtaskInfo.outputMemLength, hipMemcpyDeviceToHost );
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipMemcpy", gFuncPtr_cudaMemcpy, &lStatus, lStatusPtr, sizeof(pmStatus), hipMemcpyDeviceToHost );
	}

	if(lInputMemCudaPtr)
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lInputMemCudaPtr );

	if(lOutputMemCudaPtr)
		SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lOutputMemCudaPtr );
	
	SAFE_EXECUTE_CUDA( mRuntimeHandle, "hipFree", gFuncPtr_cudaFree, lStatusPtr );

	return lStatus;
}

#else	// SUPPORT_CUDA
/* The below functions are there to satisfy compiler. These are never executed. */
pmStatus pmDispatcherCUDA::CountAndProbeProcessingElements()
{
	mCountCUDA = 0;
	return pmSuccess;
}

pmStatus pmDispatcherCUDA::BindToDevice(size_t pDeviceIndex)
{
	return pmSuccess;
}

std::string pmDispatcherCUDA::GetDeviceName(size_t pDeviceIndex)
{
	return std::string();
}

std::string pmDispatcherCUDA::GetDeviceDescription(size_t pDeviceIndex)
{
	return std::string();
}

pmStatus pmDispatcherCUDA::InvokeKernel(pmTaskInfo& pTaskInfo, pmSubtaskInfo& pSubtaskInfo, pmSubtaskCallback_GPU_CUDA pKernelPtr)
{
	return pmSuccess;
}

#endif	// SUPPORT_CUDA

}
