
#include <hip/hip_runtime.h>

#ifdef BUILD_CUDA

#include "pmPublicDefinitions.h"
#include "commonAPI.h"
#include "fft.h"

#include <cufft.h>

#include <map>

namespace fft
{

#define CUFFT_ERROR_CHECK(name, x) \
{ \
    cufftResult dResult = x; \
    if(dResult != CUFFT_SUCCESS) \
    { \
        std::cout << name << " failed with error " << dResult << std::endl; \
        exit(1); \
    } \
}

struct cufftWrapper
{
    cufftWrapper()
    {}
    
    ~cufftWrapper()
    {
        std::map<int, cufftHandle>::iterator lIter = cufftMap.begin(), lEndIter = cufftMap.end();
        
        for(; lIter != lEndIter; ++lIter)
            CUFFT_ERROR_CHECK("cufftDestroy", cufftDestroy(lIter->second));
        
        cufftMap.clear();
    }

    std::map<int, cufftHandle> cufftMap;  // deviceId versus cufftPlan1d handle
};
    
// Assumes same pElemsY value is passed for every invocation
cufftHandle getCufftPlan1d(size_t pElemsY)
{
    static cufftWrapper lWrapper;

    int lDeviceId;
    CUDA_ERROR_CHECK("cudaGetDevice", cudaGetDevice(&lDeviceId));

    std::map<int, cufftHandle>::iterator lIter = lWrapper.cufftMap.find(lDeviceId);
    if(lIter == lWrapper.cufftMap.end())
    {
        cufftHandle lPlan;

        CUFFT_ERROR_CHECK("cufftPlan1d", cufftPlan1d(&lPlan, pElemsY, CUFFT_C2C, ROWS_PER_FFT_SUBTASK));

        cudaDeviceProp lDeviceProp;
        CUDA_ERROR_CHECK("cudaGetDeviceProperties", cudaGetDeviceProperties(&lDeviceProp, lDeviceId));
    
        lWrapper.cufftMap[lDeviceId] = lPlan;
        
        return lPlan;
    }
    
    return lIter->second;
}

pmStatus fft_cudaLaunchFunc(pmTaskInfo pTaskInfo, pmDeviceInfo pDeviceInfo, pmSubtaskInfo pSubtaskInfo, void* pCudaStream)
{
	fftTaskConf* lTaskConf = (fftTaskConf*)(pTaskInfo.taskConf);

    cufftHandle lPlan = getCufftPlan1d(lTaskConf->elemsY);
    CUFFT_ERROR_CHECK("cufftSetStream", cufftSetStream(lPlan, (cudaStream_t)pCudaStream));
    
    if(lTaskConf->inplace)
    {
        CUFFT_ERROR_CHECK("cufftExecC2C", cufftExecC2C(lPlan, (cufftComplex*)pSubtaskInfo.memInfo[INPLACE_MEM_INDEX].ptr, (cufftComplex*)pSubtaskInfo.memInfo[INPLACE_MEM_INDEX].ptr, CUFFT_FORWARD));
    }
    else
    {
        CUFFT_ERROR_CHECK("cufftExecC2C", cufftExecC2C(lPlan, (cufftComplex*)pSubtaskInfo.memInfo[INPUT_MEM_INDEX].ptr, (cufftComplex*)pSubtaskInfo.memInfo[OUTPUT_MEM_INDEX].ptr, CUFFT_FORWARD));
    }

    return pmSuccess;
}

// Returns 0 on success; non-zero on failure
int fftSingleGpu2D(bool inplace, complex* inputData, complex* outputData, size_t powx, size_t nx, size_t powy, size_t ny, int dir)
{
    void* lInputData = (inplace ? outputData : inputData);
    
    void* lInputMemCudaPtr = NULL;
    size_t lSize = sizeof(FFT_DATA_TYPE) * nx * ny;
    CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lInputMemCudaPtr, lSize));
    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(lInputMemCudaPtr, lInputData, lSize, cudaMemcpyHostToDevice));

    void* lOutputMemCudaPtr = lInputMemCudaPtr;
    if(!inplace)
        CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lOutputMemCudaPtr, lSize));
    
    cufftHandle lPlan;
    
#ifdef FFT_2D
    CUFFT_ERROR_CHECK("cufftPlan2d", cufftPlan2d(&lPlan, ny, nx, CUFFT_C2C));
#else
    CUFFT_ERROR_CHECK("cufftPlan1d", cufftPlan1d(&lPlan, ny, CUFFT_C2C, nx));
#endif
    
    CUFFT_ERROR_CHECK("cufftExecC2C", cufftExecC2C(lPlan, (cufftComplex*)lInputMemCudaPtr, (cufftComplex*)lOutputMemCudaPtr, CUFFT_FORWARD));
    CUFFT_ERROR_CHECK("cufftDestroy", cufftDestroy(lPlan));

    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(outputData, lOutputMemCudaPtr, lSize, cudaMemcpyDeviceToHost));
    CUDA_ERROR_CHECK("cudaFree", cudaFree(lOutputMemCudaPtr));
    
    if(!inplace)
        CUDA_ERROR_CHECK("cudaFree", cudaFree(lInputMemCudaPtr));
    
    return 0;
}

}

#endif