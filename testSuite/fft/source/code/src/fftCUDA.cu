
#include <hip/hip_runtime.h>

#ifdef BUILD_CUDA

#include "pmPublicDefinitions.h"
#include "commonAPI.h"
#include "fft.h"

#include <cufft.h>

#include <map>

namespace fft
{

#define CUFFT_ERROR_CHECK(name, x) \
{ \
    cufftResult dResult = x; \
    if(dResult != CUFFT_SUCCESS) \
    { \
        std::cout << name << " failed with error " << dResult << std::endl; \
        exit(1); \
    } \
}

struct cufftWrapper
{
    cufftWrapper()
    {}
    
    ~cufftWrapper()
    {
        std::map<std::pair<int, size_t>, cufftHandle>::iterator lIter = cufftMap.begin(), lEndIter = cufftMap.end();
        
        for(; lIter != lEndIter; ++lIter)
            CUFFT_ERROR_CHECK("cufftDestroy", cufftDestroy(lIter->second));
        
        cufftMap.clear();
    }

    std::map<std::pair<int, size_t>, cufftHandle> cufftMap;  // pair<deviceId, transformSize> versus cufftPlan1d handle
};

struct cufftManyWrapper
{
    cufftManyWrapper()
    {}
    
    ~cufftManyWrapper()
    {
        std::map<std::pair<int, std::pair<size_t, size_t> >, cufftHandle>::iterator lIter = cufftMap.begin(), lEndIter = cufftMap.end();
        
        for(; lIter != lEndIter; ++lIter)
            CUFFT_ERROR_CHECK("cufftDestroy", cufftDestroy(lIter->second));
        
        cufftMap.clear();
    }

    std::map<std::pair<int, std::pair<size_t, size_t> >, cufftHandle> cufftMap;  // pair<deviceId, pair<transformSize, rowStride>> versus cufftPlan1d handle
};
    
cufftHandle getCufftPlan1d(size_t pElemsY)
{
    static cufftWrapper lWrapper;

    int lDeviceId;
    CUDA_ERROR_CHECK("cudaGetDevice", cudaGetDevice(&lDeviceId));

    std::pair<int, size_t> lPair(lDeviceId, pElemsY);
    std::map<std::pair<int, size_t>, cufftHandle>::iterator lIter = lWrapper.cufftMap.find(lPair);

    if(lIter == lWrapper.cufftMap.end())
    {
        cufftHandle lPlan;

        CUFFT_ERROR_CHECK("cufftPlan1d", cufftPlan1d(&lPlan, pElemsY, CUFFT_C2C, ROWS_PER_FFT_SUBTASK));

        cudaDeviceProp lDeviceProp;
        CUDA_ERROR_CHECK("cudaGetDeviceProperties", cudaGetDeviceProperties(&lDeviceProp, lDeviceId));
    
        lWrapper.cufftMap[lPair] = lPlan;
        
        return lPlan;
    }
    
    return lIter->second;
}

cufftHandle getCufftPlanMany(size_t pN, size_t pM)
{
    static cufftManyWrapper lWrapper;

    int lDeviceId;
    CUDA_ERROR_CHECK("cudaGetDevice", cudaGetDevice(&lDeviceId));

    std::pair<int, std::pair<size_t, size_t> > lPair(lDeviceId, std::make_pair(pN, pM));
    std::map<std::pair<int, std::pair<size_t, size_t> >, cufftHandle>::iterator lIter = lWrapper.cufftMap.find(lPair);

    if(lIter == lWrapper.cufftMap.end())
    {
        cufftHandle lPlan;
        
        int lN[] = {pN};

        CUFFT_ERROR_CHECK("cufftPlanMany", cufftPlanMany(&lPlan, 1, lN, lN, (int)pM, 1, lN, (int)pM, 1, CUFFT_C2C, ROWS_PER_FFT_SUBTASK));

        cudaDeviceProp lDeviceProp;
        CUDA_ERROR_CHECK("cudaGetDeviceProperties", cudaGetDeviceProperties(&lDeviceProp, lDeviceId));
    
        lWrapper.cufftMap[lPair] = lPlan;
        
        return lPlan;
    }
    
    return lIter->second;
}

pmStatus fft_cudaLaunchFunc(pmTaskInfo pTaskInfo, pmDeviceInfo pDeviceInfo, pmSubtaskInfo pSubtaskInfo, void* pCudaStream)
{
	fftTaskConf* lTaskConf = (fftTaskConf*)(pTaskInfo.taskConf);

#ifdef NO_MATRIX_TRANSPOSE
    cufftHandle lPlan = lTaskConf->rowPlanner ? getCufftPlan1d(lTaskConf->elemsY) : getCufftPlanMany(lTaskConf->elemsX, ROWS_PER_FFT_SUBTASK);
#else
    cufftHandle lPlan = getCufftPlan1d(lTaskConf->elemsY);
#endif

    CUFFT_ERROR_CHECK("cufftSetStream", cufftSetStream(lPlan, (cudaStream_t)pCudaStream));
    
    if(lTaskConf->inplace)
    {
        CUFFT_ERROR_CHECK("cufftExecC2C", cufftExecC2C(lPlan, (cufftComplex*)pSubtaskInfo.memInfo[INPLACE_MEM_INDEX].ptr, (cufftComplex*)pSubtaskInfo.memInfo[INPLACE_MEM_INDEX].ptr, CUFFT_FORWARD));
    }
    else
    {
        CUFFT_ERROR_CHECK("cufftExecC2C", cufftExecC2C(lPlan, (cufftComplex*)pSubtaskInfo.memInfo[INPUT_MEM_INDEX].ptr, (cufftComplex*)pSubtaskInfo.memInfo[OUTPUT_MEM_INDEX].ptr, CUFFT_FORWARD));
    }

    return pmSuccess;
}

// Returns 0 on success; non-zero on failure
int fftSingleGpu2D(bool inplace, complex* inputData, complex* outputData, size_t powx, size_t nx, size_t powy, size_t ny, int dir)
{
    void* lInputData = (inplace ? outputData : inputData);
    
    void* lInputMemCudaPtr = NULL;
    size_t lSize = sizeof(FFT_DATA_TYPE) * nx * ny;
    CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lInputMemCudaPtr, lSize));
    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(lInputMemCudaPtr, lInputData, lSize, cudaMemcpyHostToDevice));

    void* lOutputMemCudaPtr = lInputMemCudaPtr;
    if(!inplace)
        CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lOutputMemCudaPtr, lSize));
    
    cufftHandle lPlan;
    
#ifdef FFT_2D
    CUFFT_ERROR_CHECK("cufftPlan2d", cufftPlan2d(&lPlan, ny, nx, CUFFT_C2C));
#else
    CUFFT_ERROR_CHECK("cufftPlan1d", cufftPlan1d(&lPlan, ny, CUFFT_C2C, nx));
#endif
    
    CUFFT_ERROR_CHECK("cufftExecC2C", cufftExecC2C(lPlan, (cufftComplex*)lInputMemCudaPtr, (cufftComplex*)lOutputMemCudaPtr, CUFFT_FORWARD));
    CUFFT_ERROR_CHECK("cufftDestroy", cufftDestroy(lPlan));

    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(outputData, lOutputMemCudaPtr, lSize, cudaMemcpyDeviceToHost));
    CUDA_ERROR_CHECK("cudaFree", cudaFree(lOutputMemCudaPtr));
    
    if(!inplace)
        CUDA_ERROR_CHECK("cudaFree", cudaFree(lInputMemCudaPtr));
    
    return 0;
}

}

#endif