
#include <hip/hip_runtime.h>

#ifdef BUILD_CUDA

#include "pmPublicDefinitions.h"
#include "matrixMultiplyBlas.h"
#include "commonAPI.h"

#include <iostream>

namespace matrixMultiplyBlas
{

#if defined(MATRIX_DATA_TYPE_FLOAT)
#define CUBLAS_GEMM cublasSgemm
#elif defined(MATRIX_DATA_TYPE_DOUBLE)
#define CUBLAS_GEMM cublasDgemm
#endif

const MATRIX_DATA_TYPE gZero = (MATRIX_DATA_TYPE)0.0;
const MATRIX_DATA_TYPE gOne = (MATRIX_DATA_TYPE)1.0;

pmStatus matrixMultiply_cudaLaunchFunc(pmTaskInfo pTaskInfo, pmDeviceInfo pDeviceInfo, pmSubtaskInfo pSubtaskInfo, void* pCudaStream)
{
    cublasHandle_t lCublasHandle = GetCublasHandle(pDeviceInfo.deviceHandle);
	matrixMultiplyTaskConf* lTaskConf = (matrixMultiplyTaskConf*)(pTaskInfo.taskConf);

    size_t lBlocksPerDim = (lTaskConf->matrixDim / lTaskConf->blockDim);
    size_t lBlockRow = (pSubtaskInfo.subtaskId / lBlocksPerDim);
    size_t lBlockCol = (pSubtaskInfo.subtaskId % lBlocksPerDim);

    size_t lBlockOffset, lBlockWidth;
    if(!GetSplitData(&lBlockOffset, &lBlockWidth, lTaskConf, pSubtaskInfo.splitInfo))
        return pmSuccess;

    MATRIX_DATA_TYPE* lMatrix1 = (MATRIX_DATA_TYPE*)(pSubtaskInfo.memInfo[INPUT_MATRIX1_MEM_INDEX].memHandle);
    MATRIX_DATA_TYPE* lMatrix2 = (MATRIX_DATA_TYPE*)(pSubtaskInfo.memInfo[INPUT_MATRIX2_MEM_INDEX].memHandle);
    MATRIX_DATA_TYPE* lMatrix3 = (MATRIX_DATA_TYPE*)(pSubtaskInfo.memInfo[OUTPUT_MATRIX_MEM_INDEX].memHandle);

    CUBLAS_ERROR_CHECK("cublasSetStream", cublasSetStream(lCublasHandle, (cudaStream_t)pCudaStream));

    CUBLAS_ERROR_CHECK("cublasSetPointerMode", cublasSetPointerMode(lCublasHandle, CUBLAS_POINTER_MODE_HOST));

    CUBLAS_ERROR_CHECK("cublas_gemm", CUBLAS_GEMM(lCublasHandle, CUBLAS_OP_N, CUBLAS_OP_N, (int)lTaskConf->blockDim, (int)lBlockWidth, (int)lTaskConf->matrixDim, &gOne, lMatrix2, (int)lTaskConf->matrixDim, lMatrix1, (int)lTaskConf->matrixDim, &gZero, lMatrix3, (int)lTaskConf->matrixDim));

    return pmSuccess;
}
    
// Returns 0 on success; non-zero on failure
int singleGpuMatrixMultiply(MATRIX_DATA_TYPE* pInputMatrices, MATRIX_DATA_TYPE* pOutputMatrix, int pDim)
{
    cublasHandleManager lManager;
    cublasHandle_t lCublasHandle = lManager.GetHandle();

    void* lInputMemCudaPtr = NULL;
    void* lOutputMemCudaPtr = NULL;

    size_t lOutputSize = sizeof(MATRIX_DATA_TYPE) * pDim * pDim;
    size_t lInputSize = 2 * lOutputSize;

    CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lInputMemCudaPtr, lInputSize));
    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(lInputMemCudaPtr, pInputMatrices, lInputSize, cudaMemcpyHostToDevice));
    CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lOutputMemCudaPtr, lOutputSize));
    
    MATRIX_DATA_TYPE* lMatrix1 = (MATRIX_DATA_TYPE*)lInputMemCudaPtr;
    MATRIX_DATA_TYPE* lMatrix2 = lMatrix1 + pDim * pDim;
    MATRIX_DATA_TYPE* lMatrix3 = (MATRIX_DATA_TYPE*)lOutputMemCudaPtr;

    CUBLAS_ERROR_CHECK("cublasSetPointerMode", cublasSetPointerMode(lCublasHandle, CUBLAS_POINTER_MODE_HOST));

    CUBLAS_ERROR_CHECK("cublas_gemm", CUBLAS_GEMM(lCublasHandle, CUBLAS_OP_N, CUBLAS_OP_N, pDim, pDim, pDim, &gOne, lMatrix2, pDim, lMatrix1, pDim, &gZero, lMatrix3, pDim));
    
    CUDA_ERROR_CHECK("cudaDeviceSynchronize", cudaDeviceSynchronize());

    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(pOutputMatrix, lOutputMemCudaPtr, lOutputSize, cudaMemcpyDeviceToHost));
    CUDA_ERROR_CHECK("cudaFree", cudaFree(lOutputMemCudaPtr));
    CUDA_ERROR_CHECK("cudaFree", cudaFree(lInputMemCudaPtr));

    return 0;
}

}

#endif