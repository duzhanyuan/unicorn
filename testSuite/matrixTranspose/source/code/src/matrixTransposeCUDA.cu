#include "hip/hip_runtime.h"

#include "pmPublicDefinitions.h"
#include "commonAPI.h"
#include "matrixTranspose.h"

#include <iostream>

#ifdef BUILD_CUDA
#ifdef USE_SQUARE_BLOCKS

namespace matrixTranspose
{

__global__ void matrixTranspose_cuda(size_t pInputMemCols, size_t pSubtaskRows, void* pInputMem, void* pOutputBlock)
{
    __shared__ MATRIX_DATA_TYPE lTile[GPU_TILE_DIM][GPU_TILE_DIM + 1];

    int lBlockId_x = (blockIdx.x + blockIdx.y) % gridDim.x;
    int lBlockId_y = blockIdx.x;
    
    int lIndexX = lBlockId_x * GPU_TILE_DIM + threadIdx.x;
    int lIndexY = lBlockId_y * GPU_TILE_DIM + threadIdx.y;
    int lInputIndex = lIndexX + (lIndexY * pInputMemCols);

    lIndexX = lBlockId_y * GPU_TILE_DIM + threadIdx.x;
    lIndexY = lBlockId_x * GPU_TILE_DIM + threadIdx.y;
    int lOutputIndex = lIndexX + (lIndexY * pSubtaskRows);

    int i, lStride = (GPU_TILE_DIM/GPU_ELEMS_PER_THREAD);
    for(i = 0; i < GPU_TILE_DIM; i += lStride)
        lTile[threadIdx.y + i][threadIdx.x] = ((MATRIX_DATA_TYPE*)pInputMem)[lInputIndex + i * pInputMemCols];

    __syncthreads();

    for(i = 0; i < GPU_TILE_DIM; i += lStride)
        ((MATRIX_DATA_TYPE*)pOutputBlock)[lOutputIndex + i * pSubtaskRows] = lTile[threadIdx.x][threadIdx.y + i];
}

__global__ void matrixCopy_cuda(matrixTransposeTaskConf pTaskConf, pmSubtaskInfo pSubtaskInfo, void* pOutputBlock)
{
    int lIndexX = blockIdx.x * GPU_TILE_DIM + threadIdx.x;
    int lIndexY = blockIdx.y * GPU_TILE_DIM + threadIdx.y;
    int lInputIndex = lIndexX + (lIndexY * pTaskConf.blockSizeRows);
    int lOutputIndex = lIndexX + (lIndexY * pTaskConf.matrixDimRows);
    
    unsigned int lOutputMemIndex = (lTaskConf->inplace ? INPLACE_MEM_INDEX : OUTPUT_MEM_INDEX);

    int i, lStride = (GPU_TILE_DIM/GPU_ELEMS_PER_THREAD);
    for(i = 0; i < GPU_TILE_DIM; i += lStride)
        ((MATRIX_DATA_TYPE*)pSubtaskInfo.memInfo[lOutputMemIndex].writePtr)[lOutputIndex + i * pTaskConf.matrixDimRows] = ((MATRIX_DATA_TYPE*)pOutputBlock)[lInputIndex + i * pTaskConf.blockSizeRows];
}
    
__global__ void matrixTranspose_singleGpu(size_t pInputMemCols, size_t pSubtaskRows, void* pInputMem, void* pOutputBlock, size_t pMaxBlocksX, size_t pMaxBlocksY)
{
    __shared__ MATRIX_DATA_TYPE lTile[GPU_TILE_DIM][GPU_TILE_DIM + 1];

    int lBlockId_x = (blockIdx.x + blockIdx.y) % gridDim.x;
    int lBlockId_y = blockIdx.x;
    
    if(lBlockId_x >= pMaxBlocksX || lBlockId_y >= pMaxBlocksY)
        return;
    
    int lIndexX = lBlockId_x * GPU_TILE_DIM + threadIdx.x;
    int lIndexY = lBlockId_y * GPU_TILE_DIM + threadIdx.y;
    int lInputIndex = lIndexX + (lIndexY * pInputMemCols);

    lIndexX = lBlockId_y * GPU_TILE_DIM + threadIdx.x;
    lIndexY = lBlockId_x * GPU_TILE_DIM + threadIdx.y;
    int lOutputIndex = lIndexX + (lIndexY * pSubtaskRows);

    int i, lStride = (GPU_TILE_DIM/GPU_ELEMS_PER_THREAD);
    for(i = 0; i < GPU_TILE_DIM; i += lStride)
        lTile[threadIdx.y + i][threadIdx.x] = ((MATRIX_DATA_TYPE*)pInputMem)[lInputIndex + i * pInputMemCols];

    __syncthreads();

    for(i = 0; i < GPU_TILE_DIM; i += lStride)
        ((MATRIX_DATA_TYPE*)pOutputBlock)[lOutputIndex + i * pSubtaskRows] = lTile[threadIdx.x][threadIdx.y + i];
}

pmStatus matrixTranspose_cudaLaunchFunc(pmTaskInfo pTaskInfo, pmDeviceInfo pDeviceInfo, pmSubtaskInfo pSubtaskInfo, void* pCudaStream)
{
	matrixTransposeTaskConf* lTaskConf = (matrixTransposeTaskConf*)(pTaskInfo.taskConf);

    dim3 gridConf(lTaskConf->blockSizeRows / GPU_TILE_DIM, lTaskConf->blockSizeRows / GPU_TILE_DIM, 1);
    dim3 blockConf(GPU_TILE_DIM, GPU_TILE_DIM / GPU_ELEMS_PER_THREAD, 1);

    hipStream_t lCudaStream = (hipStream_t)pCudaStream;
    
    if(lTaskConf->inplace)
    {
        void* lBlockCudaPtr = pSubtaskInfo.gpuContext.reservedGlobalMem;

        matrixTranspose_cuda <<<gridConf, blockConf, 0, lCudaStream>>> (lTaskConf->matrixDimCols, lTaskConf->blockSizeRows, pSubtaskInfo.memInfo[OUTPUT_MEM_INDEX].readPtr, lBlockCudaPtr);
        matrixCopy_cuda <<<gridConf, blockConf, 0, lCudaStream>>> (*lTaskConf, pSubtaskInfo, lBlockCudaPtr);    // because transpose is inplace, this has to be a post step
    }
    else
    {
        matrixTranspose_cuda <<<gridConf, blockConf, 0, lCudaStream>>> (lTaskConf->matrixDimCols, lTaskConf->matrixDimRows, pSubtaskInf.memInfo[INPUT_MEM_INDEX].ptr, pSubtaskInfo.memInfo[OUTPUT_MEM_INDEX].ptr);
    }
    
    return pmSuccess;
}
    
int singleGpuMatrixTranspose(bool pInplace, MATRIX_DATA_TYPE* pInputMatrix, MATRIX_DATA_TYPE* pOutputMatrix, size_t pInputDimRows, size_t pInputDimCols)
{
    MATRIX_DATA_TYPE* lInputMatrix = pInplace ? pOutputMatrix : pInputMatrix;
    
    void* lInputMemCudaPtr = NULL;
    void* lOutputMemCudaPtr = NULL;
    
    size_t lSize = sizeof(MATRIX_DATA_TYPE) * pInputDimRows * pInputDimCols;

    CUDA_ERROR_CHECK("hipMalloc", hipMalloc((void**)&lInputMemCudaPtr, lSize));
    CUDA_ERROR_CHECK("hipMemcpy", hipMemcpy(lInputMemCudaPtr, lInputMatrix, lSize, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK("hipMalloc", hipMalloc((void**)&lOutputMemCudaPtr, lSize));

    size_t lGridDim = std::max(pInputDimRows / GPU_TILE_DIM, pInputDimCols / GPU_TILE_DIM);

    dim3 gridConf(lGridDim, lGridDim, 1);
    dim3 blockConf(GPU_TILE_DIM, GPU_TILE_DIM / GPU_ELEMS_PER_THREAD, 1);
    matrixTranspose_singleGpu <<<gridConf, blockConf>>> (pInputDimCols, pInputDimRows, lInputMemCudaPtr, lOutputMemCudaPtr, pInputDimCols / GPU_TILE_DIM, pInputDimRows / GPU_TILE_DIM);

    CUDA_ERROR_CHECK("hipDeviceSynchronize", hipDeviceSynchronize());

    CUDA_ERROR_CHECK("hipMemcpy", hipMemcpy(pOutputMatrix, lOutputMemCudaPtr, lSize, hipMemcpyDeviceToHost));
    CUDA_ERROR_CHECK("hipFree", hipFree(lInputMemCudaPtr));
    CUDA_ERROR_CHECK("hipFree", hipFree(lOutputMemCudaPtr));
    
    return 0;
}

}

#endif
#endif

