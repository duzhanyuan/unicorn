
#include <hip/hip_runtime.h>

/*
 * Copyright (c) 2016, Tarun Beri, Sorav Bansal, Subodh Kumar
 * Copyright (c) 2016 Indian Institute of Technology Delhi
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version. Any redistribution or
 * modification must retain this copyright notice and appropriately
 * highlight the credits.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA
 *
 * More information about the authors is available at their websites -
 * Prof. Subodh Kumar - http://www.cse.iitd.ernet.in/~subodh/
 * Prof. Sorav Bansal - http://www.cse.iitd.ernet.in/~sbansal/
 * Tarun Beri - http://www.cse.iitd.ernet.in/~tarun
 *
 * All bug reports and enhancement requests can be sent to the following
 * email addresses -
 * onlinetarun@gmail.com
 * sbansal@cse.iitd.ac.in
 * subodh@cse.iitd.ac.in
 */

#ifdef BUILD_CUDA

#include "pmPublicDefinitions.h"
#include "pmPublicUtilities.h"
#include "commonAPI.h"
#include "pageRank.h"

#include <cmath>

namespace pageRank
{
    
__global__ void pageRank_cuda(pageRankTaskConf pTaskConf, unsigned int pWebPages, PAGE_RANK_DATA_TYPE* pLocalArray, PAGE_RANK_DATA_TYPE* pGlobalArray, unsigned int* pSubtaskWebDump)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
	int threadId = blockId * blockDim.x + threadIdx.x;

	if(threadId >= pWebPages)
		return;

	unsigned int index = threadId * (1 + pTaskConf.maxOutlinksPerWebPage);
    unsigned int outlinks = pSubtaskWebDump[index++];
    PAGE_RANK_DATA_TYPE lIncr = (PAGE_RANK_DATA_TYPE)(DAMPENING_FACTOR * ((pTaskConf.iteration == 0) ? pTaskConf.initialPageRank : pLocalArray[threadId])/(float)outlinks);

    for(unsigned int k = 0; k < outlinks; ++k)
    {
        unsigned int lRefLink = pSubtaskWebDump[index + k];
        PAGE_RANK_DATA_TYPE* lAddress = (PAGE_RANK_DATA_TYPE*)(pGlobalArray + lRefLink - 1);
        
        atomicAdd(lAddress, lIncr);
    }
}
    
__global__ void zeroInit(PAGE_RANK_DATA_TYPE* pGlobalArray, unsigned int pWebPages)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
	int threadId = blockId * blockDim.x + threadIdx.x;
    
	if(threadId >= pWebPages)
		return;

    pGlobalArray[threadId] = 0;
}

pmCudaLaunchConf GetCudaLaunchConf(unsigned int pWebPages)
{
    const unsigned int maxThreadsPerBlockDim = 512;
    const unsigned int maxBlocksPerGridDim = 65535;
    pmCudaLaunchConf lCudaLaunchConf;
    
    if(pWebPages > maxThreadsPerBlockDim)
    {
        lCudaLaunchConf.blocksX = pWebPages / maxThreadsPerBlockDim + ((pWebPages % maxThreadsPerBlockDim) ? 1 : 0);
        lCudaLaunchConf.threadsX = maxThreadsPerBlockDim;
    }
    else
    {
        lCudaLaunchConf.blocksX = 1;
        lCudaLaunchConf.threadsX = pWebPages;
    }
    
    lCudaLaunchConf.blocksY = 1;
    
    if(lCudaLaunchConf.blocksX > maxBlocksPerGridDim)
    {
        double lSqrt = std::sqrt((double)lCudaLaunchConf.blocksX);
        unsigned int lFloor = (unsigned int)std::floor(lSqrt);
        unsigned int lCeil = lFloor;

        if(lFloor * lFloor < lCudaLaunchConf.blocksX)
            lCeil = lFloor + 1;
        
        if(lCeil * lFloor < lCudaLaunchConf.blocksX)
            lFloor = lCeil;
        
        if(lCeil * lFloor < lCudaLaunchConf.blocksX)
            exit(1);

        lCudaLaunchConf.blocksX = lCeil;
        lCudaLaunchConf.blocksY = lFloor;
    }
    
    return lCudaLaunchConf;
}
    
pmStatus pageRank_cudaLaunchFunc(pmTaskInfo pTaskInfo, pmDeviceInfo pDeviceInfo, pmSubtaskInfo pSubtaskInfo, void* pCudaStream)
{
    pageRankTaskConf* lTaskConf = (pageRankTaskConf*)(pTaskInfo.taskConf);
    
    ulong lSubtaskId = pSubtaskInfo.subtaskId;
    void** lWebFilePtrs = LoadMappedFiles(lTaskConf, lSubtaskId);
    
    unsigned int lWebPages = (unsigned int)((lTaskConf->totalWebPages < ((lSubtaskId + 1) * lTaskConf->webPagesPerSubtask)) ? (lTaskConf->totalWebPages - (lSubtaskId * lTaskConf->webPagesPerSubtask)) : lTaskConf->webPagesPerSubtask);
    unsigned int lWebFiles = ((lWebPages / lTaskConf->webPagesPerFile) + ((lWebPages % lTaskConf->webPagesPerFile) ? 1 : 0));
    unsigned int lFirstWebFile = (unsigned int)pSubtaskInfo.subtaskId * lWebFiles;

    unsigned int* lWebDump = (unsigned int*)(pSubtaskInfo.gpuContext.reservedGlobalMem);
    unsigned int* lWebDumpPtr = lWebDump;

    unsigned int lTotalFiles = (lTaskConf->totalWebPages / lTaskConf->webPagesPerFile) + ((lTaskConf->totalWebPages % lTaskConf->webPagesPerFile) ? 1 : 0);
    for(unsigned int i = 0; i < lWebFiles; ++i)
    {
        unsigned int* lMappedFile = (unsigned int*)(lWebFilePtrs[i]);
        
        unsigned int lPagesInFile = lTaskConf->webPagesPerFile;
        if(i + lFirstWebFile == lTotalFiles - 1)
            lPagesInFile = lTaskConf->totalWebPages - (i + lFirstWebFile) * lTaskConf->webPagesPerFile;

        CUDA_ERROR_CHECK("cudaMemcpyAsync", cudaMemcpyAsync(lWebDumpPtr, lMappedFile, sizeof(unsigned int) * lPagesInFile * (lTaskConf->maxOutlinksPerWebPage + 1), cudaMemcpyHostToDevice, (cudaStream_t)pCudaStream));
        lWebDumpPtr += lPagesInFile * (lTaskConf->maxOutlinksPerWebPage + 1);
    }
    
    delete[] lWebFilePtrs;
    
	PAGE_RANK_DATA_TYPE* lLocalArray = ((lTaskConf->iteration == 0) ? NULL : (PAGE_RANK_DATA_TYPE*)pSubtaskInfo.memInfo[INPUT_MEM_INDEX].ptr);
    PAGE_RANK_DATA_TYPE* lGlobalArray = (PAGE_RANK_DATA_TYPE*)pSubtaskInfo.memInfo[OUTPUT_MEM_INDEX].ptr;

    pmCudaLaunchConf lGlobalCudaLaunchConf = GetCudaLaunchConf(lTaskConf->totalWebPages);
    dim3 globalGridConf(lGlobalCudaLaunchConf.blocksX, lGlobalCudaLaunchConf.blocksY, 1);
    dim3 globalBlockConf(lGlobalCudaLaunchConf.threadsX, 1, 1);

    zeroInit<<<globalGridConf, globalBlockConf>>>(lGlobalArray, lTaskConf->totalWebPages);

    pmCudaLaunchConf lCudaLaunchConf = GetCudaLaunchConf(lWebPages);
    dim3 gridConf(lCudaLaunchConf.blocksX, lCudaLaunchConf.blocksY, 1);
    dim3 blockConf(lCudaLaunchConf.threadsX, 1, 1);

    pageRank_cuda<<<gridConf, blockConf, 0, (cudaStream_t)pCudaStream>>>(*lTaskConf, lWebPages, lLocalArray, lGlobalArray, lWebDump);
    
    return pmSuccess;
}
    
// Returns 0 on success, non-zero on failure
int singleGpuPageRank(pageRankTaskConf& pTaskConf, unsigned int* pWebDump, void* pOutputMem)
{
    unsigned int* lWebDumpCudaPtr = NULL;
    PAGE_RANK_DATA_TYPE* lLocalArrayCudaPtr = NULL;
    PAGE_RANK_DATA_TYPE* lGlobalArrayCudaPtr = NULL;

    size_t lOutputSize = pTaskConf.totalWebPages * sizeof(PAGE_RANK_DATA_TYPE);
    size_t lWebSize = pTaskConf.totalWebPages * (pTaskConf.maxOutlinksPerWebPage + 1) * sizeof(unsigned int);

    CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lWebDumpCudaPtr, lWebSize));
    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(lWebDumpCudaPtr, pWebDump, lWebSize, cudaMemcpyHostToDevice));
    
    pmCudaLaunchConf lCudaLaunchConf = GetCudaLaunchConf(pTaskConf.totalWebPages);
    dim3 gridConf(lCudaLaunchConf.blocksX, 1, 1);
    dim3 blockConf(lCudaLaunchConf.threadsX, 1, 1);

    for(unsigned int i = 0; i < PAGE_RANK_ITERATIONS; ++i)
    {
		if(i > 1)
            CUDA_ERROR_CHECK("cudaFree", cudaFree(lLocalArrayCudaPtr));

        lLocalArrayCudaPtr = lGlobalArrayCudaPtr;
        
        CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lGlobalArrayCudaPtr, lOutputSize));

        pTaskConf.iteration = i;
        
        zeroInit<<<gridConf, blockConf>>>(lGlobalArrayCudaPtr, pTaskConf.totalWebPages);
        pageRank_cuda<<<gridConf, blockConf>>>(pTaskConf, pTaskConf.totalWebPages, lLocalArrayCudaPtr, lGlobalArrayCudaPtr, lWebDumpCudaPtr);
    }
    
    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(pOutputMem, lGlobalArrayCudaPtr, lOutputSize, cudaMemcpyDeviceToHost));

    if(lLocalArrayCudaPtr)
        CUDA_ERROR_CHECK("cudaFree", cudaFree(lLocalArrayCudaPtr));

    CUDA_ERROR_CHECK("cudaFree", cudaFree(lWebDumpCudaPtr));
    CUDA_ERROR_CHECK("cudaFree", cudaFree(lGlobalArrayCudaPtr));

    return 0;
}

}

#endif
