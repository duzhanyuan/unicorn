
#include <hip/hip_runtime.h>

#ifdef BUILD_CUDA

#include "pmPublicDefinitions.h"
#include "pmPublicUtilities.h"
#include "commonAPI.h"
#include "pageRank.h"

namespace pageRank
{
    
__global__ void pageRank_cuda(pageRankTaskConf pTaskConf, unsigned int pWebPages, PAGE_RANK_DATA_TYPE* pLocalArray, PAGE_RANK_DATA_TYPE* pGlobalArray, unsigned int* pSubtaskWebDump)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadId >= pWebPages)
		return;

	unsigned int index = threadId * (1 + pTaskConf.maxOutlinksPerWebPage);
    unsigned int outlinks = pSubtaskWebDump[index++];
    PAGE_RANK_DATA_TYPE lIncr = (PAGE_RANK_DATA_TYPE)(DAMPENING_FACTOR * ((pTaskConf.iteration == 0) ? pTaskConf.initialPageRank : pLocalArray[threadId])/(float)outlinks);

    for(unsigned int k = 0; k < outlinks; ++k)
    {
        unsigned int lRefLink = pSubtaskWebDump[index + k];
        PAGE_RANK_DATA_TYPE* lAddress = (PAGE_RANK_DATA_TYPE*)(pGlobalArray + lRefLink - 1);
        
        atomicAdd(lAddress, lIncr);
    }
}

__global__ void pageRank_cuda_key_val_generator(pageRankTaskConf pTaskConf, unsigned int pWebPages, PAGE_RANK_DATA_TYPE* pLocalArray, char* pScratchBuffer, unsigned int* pSubtaskWebDump)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadId >= pWebPages)
		return;

    /* The format of storage for every page (input to this subtask) in scratch buffer is -
     * <No. of outlinks> <Incr> <Key 1> <Key 2> ... <Key (no. of outlinks)>
     */

    // Value for every key here is same (i.e. input page rank / outlinks); so no need to store key value pairs
    size_t lPerPageStorageSize = sizeof(unsigned int) + sizeof(PAGE_RANK_DATA_TYPE) + pTaskConf.maxOutlinksPerWebPage * sizeof(unsigned int);   // no. of outlinks, value (i.e. increment) and keys
    size_t lScratchBufferWriteOffsetInBytes = threadId * lPerPageStorageSize;

	unsigned int index = threadId * (1 + pTaskConf.maxOutlinksPerWebPage);
    unsigned int lOutlinks = pSubtaskWebDump[index++];
    PAGE_RANK_DATA_TYPE lIncr = (PAGE_RANK_DATA_TYPE)(DAMPENING_FACTOR * ((pTaskConf.iteration == 0) ? pTaskConf.initialPageRank : pLocalArray[threadId])/(float)lOutlinks);

    char* lWriteLocation = (char*)pScratchBuffer + lScratchBufferWriteOffsetInBytes;
    
    ((unsigned int*)lWriteLocation)[0] = lOutlinks;
    ((PAGE_RANK_DATA_TYPE*)(lWriteLocation + sizeof(unsigned int)))[0] = lIncr;

    for(unsigned int k = 0; k < lOutlinks; ++k)
        ((unsigned int*)(lWriteLocation + sizeof(unsigned int) + sizeof(PAGE_RANK_DATA_TYPE) + k * sizeof(unsigned int)))[0] = pSubtaskWebDump[index + k] - 1;
}
    
__global__ void zeroInit(PAGE_RANK_DATA_TYPE* pGlobalArray, unsigned int pWebPages)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
	if(threadId >= pWebPages)
		return;

    pGlobalArray[threadId] = (PAGE_RANK_DATA_TYPE)0;
}

pmCudaLaunchConf GetCudaLaunchConf(unsigned int pWebPages)
{
    pmCudaLaunchConf lCudaLaunchConf;
    
    if(pWebPages > 512)
    {
        lCudaLaunchConf.blocksX = pWebPages/512 + ((pWebPages%512) ? 1 : 0);
        lCudaLaunchConf.threadsX = 512;
    }
    else
    {
        lCudaLaunchConf.blocksX = 1;
        lCudaLaunchConf.threadsX = pWebPages;
    }
    
    return lCudaLaunchConf;
}
    
pmStatus pageRank_cudaLaunchFunc(pmTaskInfo pTaskInfo, pmDeviceInfo pDeviceInfo, pmSubtaskInfo pSubtaskInfo, void* pCudaStream)
{
    pageRankTaskConf* lTaskConf = (pageRankTaskConf*)(pTaskInfo.taskConf);
    
    ulong lSubtaskId = pSubtaskInfo.subtaskId;
    void** lWebFilePtrs = LoadMappedFiles(pTaskInfo, pSubtaskInfo);
    
    unsigned int lWebPages = (unsigned int)((lTaskConf->totalWebPages < ((lSubtaskId + 1) * lTaskConf->webPagesPerSubtask)) ? (lTaskConf->totalWebPages - (lSubtaskId * lTaskConf->webPagesPerSubtask)) : lTaskConf->webPagesPerSubtask);
    unsigned int lWebFiles = ((lWebPages / lTaskConf->webPagesPerFile) + ((lWebPages % lTaskConf->webPagesPerFile) ? 1 : 0));
    unsigned int lFirstWebFile = (unsigned int)pSubtaskInfo.subtaskId * lWebFiles;

    unsigned int* lWebDump = (unsigned int*)(pSubtaskInfo.gpuContext.reservedGlobalMem);
    unsigned int* lWebDumpPtr = lWebDump;

    unsigned int lTotalFiles = (lTaskConf->totalWebPages / lTaskConf->webPagesPerFile) + ((lTaskConf->totalWebPages % lTaskConf->webPagesPerFile) ? 1 : 0);
    for(unsigned int i = 0; i < lWebFiles; ++i)
    {
        unsigned int* lMappedFile = (unsigned int*)(lWebFilePtrs[i]);
        
        unsigned int lPagesInFile = lTaskConf->webPagesPerFile;
        if(i + lFirstWebFile == lTotalFiles - 1)
            lPagesInFile = lTaskConf->totalWebPages - (i + lFirstWebFile) * lTaskConf->webPagesPerFile;

        CUDA_ERROR_CHECK("cudaMemcpyAsync", cudaMemcpyAsync(lWebDumpPtr, lMappedFile, sizeof(unsigned int) * lPagesInFile * (lTaskConf->maxOutlinksPerWebPage + 1), cudaMemcpyHostToDevice, (cudaStream_t)pCudaStream));
        lWebDumpPtr += lPagesInFile * (lTaskConf->maxOutlinksPerWebPage + 1);
    }
    
    delete[] lWebFilePtrs;
    
	PAGE_RANK_DATA_TYPE* lLocalArray = ((lTaskConf->iteration == 0) ? NULL : (PAGE_RANK_DATA_TYPE*)pSubtaskInfo.memInfo[MEM_INDEX].ptr);

    pmCudaLaunchConf lCudaLaunchConf = GetCudaLaunchConf(lWebPages);
    dim3 gridConf(lCudaLaunchConf.blocksX, 1, 1);
    dim3 blockConf(lCudaLaunchConf.threadsX, 1, 1);
    pageRank_cuda_key_val_generator<<<gridConf, blockConf, 0, (cudaStream_t)pCudaStream>>>(*lTaskConf, lWebPages, lLocalArray, (char*)pSubtaskInfo.gpuContext.scratchBuffer, lWebDump);
    
    return pmSuccess;
}
    
// Returns 0 on success, non-zero on failure
int singleGpuPageRank(pageRankTaskConf& pTaskConf, unsigned int* pWebDump, void* pOutputMem)
{
    unsigned int* lWebDumpCudaPtr = NULL;
    PAGE_RANK_DATA_TYPE* lLocalArrayCudaPtr = NULL;
    PAGE_RANK_DATA_TYPE* lGlobalArrayCudaPtr = NULL;

    size_t lOutputSize = pTaskConf.totalWebPages * sizeof(PAGE_RANK_DATA_TYPE);
    size_t lWebSize = pTaskConf.totalWebPages * (pTaskConf.maxOutlinksPerWebPage + 1) * sizeof(unsigned int);

    CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lWebDumpCudaPtr, lWebSize));
    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(lWebDumpCudaPtr, pWebDump, lWebSize, cudaMemcpyHostToDevice));
    
    pmCudaLaunchConf lCudaLaunchConf = GetCudaLaunchConf(pTaskConf.totalWebPages);
    dim3 gridConf(lCudaLaunchConf.blocksX, 1, 1);
    dim3 blockConf(lCudaLaunchConf.threadsX, 1, 1);

    for(unsigned int i = 0; i < PAGE_RANK_ITERATIONS; ++i)
    {
		if(i > 1)
            CUDA_ERROR_CHECK("cudaFree", cudaFree(lLocalArrayCudaPtr));

        lLocalArrayCudaPtr = lGlobalArrayCudaPtr;
        
        CUDA_ERROR_CHECK("cudaMalloc", cudaMalloc((void**)&lGlobalArrayCudaPtr, lOutputSize));

        pTaskConf.iteration = i;
        
        zeroInit<<<gridConf, blockConf>>>(lGlobalArrayCudaPtr, pTaskConf.totalWebPages);
        pageRank_cuda<<<gridConf, blockConf>>>(pTaskConf, pTaskConf.totalWebPages, lLocalArrayCudaPtr, lGlobalArrayCudaPtr, lWebDumpCudaPtr);
    }
    
    CUDA_ERROR_CHECK("cudaMemcpy", cudaMemcpy(pOutputMem, lGlobalArrayCudaPtr, lOutputSize, cudaMemcpyDeviceToHost));

    if(lLocalArrayCudaPtr)
        CUDA_ERROR_CHECK("cudaFree", cudaFree(lLocalArrayCudaPtr));

    CUDA_ERROR_CHECK("cudaFree", cudaFree(lWebDumpCudaPtr));
    CUDA_ERROR_CHECK("cudaFree", cudaFree(lGlobalArrayCudaPtr));

    return 0;
}

}

#endif
